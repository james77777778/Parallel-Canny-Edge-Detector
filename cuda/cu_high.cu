
#include <hip/hip_runtime.h>
extern "C" __global__
void cu_high(float* final_img, float* edge_img, float* strong_edge_mask,
                        float t_high, unsigned int img_height, unsigned int img_width)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < (img_height * img_width)) {
        // apply high threshold
        if (edge_img[idx] > t_high) {
            strong_edge_mask[idx] = 1;
            final_img[idx] = 1;
        } else {
            strong_edge_mask[idx] = 0;
            final_img[idx] = 0;
        }
    }
}